#include "hip/hip_runtime.h"
#include "Interpolate3DSpherical.cu"
#include "Geometry.cu"
#include <math.h>
#include <stdlib.h>

#define xres 288
#define yres 480
#define zres 144
__device__ double coordRes = 1000;

__device__ double *xVals;
__device__ double *yVals;
__device__ double *zVals;
__device__ double *xVel;
__device__ double *yVel;
__device__ double *zVel;
__device__ double planetCoords[3] = {1, M_PI, 0.5*M_PI};
__device__ double planetVel[3] = {0, 1, 0};

__device__ double* RK5(double* (*func)(double, double*), double* y0, double xi, double h, int size){
    //Allocating memory for calculating k values
    double *k1 = (double *)malloc(size*sizeof(double));
    double *k2 = (double *)malloc(size*sizeof(double));
    double *k3 = (double *)malloc(size*sizeof(double));
    double *k4 = (double *)malloc(size*sizeof(double));
    double *k5 = (double *)malloc(size*sizeof(double));
    double *k6 = (double *)malloc(size*sizeof(double));
    double *tempy = (double *)malloc(size*sizeof(double));
    double *tempk = (double *)malloc(size*sizeof(double));
    double *sol = (double *)malloc(size*sizeof(double));

    tempk = func(xi, y0);
    for(int i = 0; i < size; i++){k1[i] = h*tempk[i];}

    for(int i = 0; i < size; i++){tempy[i] = y0[i] + k1[i]/4.0;}
    tempk = func(xi + h/4.0, tempy);
	for(int i = 0; i < size; i++){k2[i] = h*tempk[i];}

	for(int i = 0; i < size; i++){tempy[i] = y0[i] + 3.0*k1[i]/32.0 + 9.0*k2[i]/32.0;}
	tempk = func(xi + 3.0*h/8.0, tempy);
	for(int i = 0; i < size; i++){k3[i] = h*tempk[i];}

	for(int i = 0; i < size; i++){tempy[i] = y0[i] + 1932.0*k1[i]/2197.0 - 7200.0*k2[i]/2197.0 + 7296.0*k3[i]/2197.0;}
	tempk = func(xi + 12.0*h/13.0, tempy);
	for(int i = 0; i < size; i++){k4[i] = h*tempk[i];}

	for(int i = 0; i < size; i++){tempy[i] = y0[i] + 439.0*k1[i]/216.0 - 8.0*k2[i] + 3680*k3[i]/513.0 - 845*k4[i]/4104.0;}
	tempk = func(xi + h, tempy);
	for(int i = 0; i < size; i++){k5[i] = h*tempk[i];}

	for(int i = 0; i < size; i++){tempy[i] = y0[i] - 8.0*k1[i]/27.0 + 2.0*k2[i] - 3544.0*k3[i]/2565.0 + 1859.0*k4[i]/4104.0 - 11.0*k5[i]/40.0;}
	tempk = func(xi + h/2.0, tempy);
	for(int i = 0; i < size; i++){k6[i] = h*tempk[i];}

	for(int i = 0; i < size; i++){sol[i] = 16.0*k1[i]/135.0 + 6656.0*k3[i]/12825.0 + 28561.0*k4[i]/56430.0 - 9.0*k5[i]/50.0 + 2.0*k6[i]/55.0;}
    return sol;
}

__device__ double absmax(double vals[], int len){
	double max = 0;
	for(int i = 0; i < len; i++){
		if(fabs(vals[i]) > max){max = fabs(vals[i]);}
	}
	return max;
}

__device__ double absmin(double vals[], int len){
	double min = 1.7e308;
	for(int i = 0; i < len; i++){
		if(vals[i] < 0){vals[i] *= -1;}
		if(vals[i] < min && vals[i] != 0){min = vals[i];}
	}
	return min;
}

__device__ double findH(double *r, double* (*func)(double, double*), double *coordX, double *coordY, double *coordZ, int direction){
	double x = r[0];
	double y = r[1];
	double z = r[2];

	double *v = func(x, r);
	double vx = v[0];
	double vy = v[1];
	double vz = v[2];

	int *xPoints = binSearch(coordX, 0, coordRes, x, coordRes);
	int *yPoints = binSearch(coordY, 0, coordRes, y, coordRes);
	int *zPoints = binSearch(coordZ, 0, coordRes, z, coordRes);
	if(xPoints[0] == -1){return 0;}
	else if(xPoints[1] == -1){return 0;}

	if(yPoints[0] == -1){
		yPoints[0] = yres-1;
		yPoints[1] = 0;
	}
	else if(yPoints[1] == -1){
		yPoints[0] = 0;
		yPoints[1] = yres - 1;
	}

	if(zPoints[0] == -1){return 0;}
	if(zPoints[1] == -1){
		if(z > M_PI/2.0){z = M_PI - z;
			zPoints = binSearch(coordZ, 0, coordRes, z, coordRes);
		if(zPoints[0] == -1 || zPoints[1] == -1){return 0;}
		}
	else{
		zPoints[0] = coordRes-1;
		zPoints[1] = coordRes-1;
	}
	}
	int xp = xPoints[direction];
	int yp = yPoints[direction];
	int zp = zPoints[direction];
	double hx, hy, hz;
	if(vx != 0){hx = (coordX[xp] - x)/vx;}
	else{hx = 0;}
	if(vy != 0){hy = (coordY[yp] - y)/vy;}
	else{hy = 0;}
	if(vz != 0){hz = (coordZ[zp] - z)/vz;}
	else{hz = 0;}
	if(hx < 0){hx *= -1;}
	if(hy < 0){hy *= -1;}
	if(hz < 0){hz *= -1;}
	if(hx == 0 && hy == 0 && hz == 0){return 0;}
	double hs[3] = {hx, hy, hz};
	double h = absmin(hs, 3);
	return h;
	}

__device__ double myabs(double x){
	if(x < 0){return -x;}
	else{return x;}
}

__device__ double **flowLine3D(double xi, double yi, double zi, double Xs[], double Ys[], double Zs[], double* (*func)(double, double*), int maxsteps){
	double xf, yf, zf, h, phi, lastPhi, phiInitial;
	double *correction;
	double **sol = (double **)malloc(3*sizeof(double *));
	int n;

	double *xs = (double *)malloc(maxsteps*sizeof(double)+1);
	double *ys = (double *)malloc(maxsteps*sizeof(double)+1);
	double *zs = (double *)malloc(maxsteps*sizeof(double)+1);


	double y0[3] = {xi, yi, zi};

	xf = absmax(Xs, coordRes);
	yf = absmax(Ys, coordRes);
	zf = absmax(Zs, coordRes);

	xs[0] = xi;
	ys[0] = yi;
	zs[0] = zi;

	n = 0;

	phiInitial = cartesianToSpherical(y0, 3)[1];
	lastPhi = phiInitial;
	h = findH(y0, func, Xs, Ys, Zs, 1);
	while((myabs(y0[0]) <= myabs(xf)) && (myabs(y0[1]) <= myabs(yf)) && (myabs(y0[2]) <= myabs(zf))){
		if(h == 0 || h > 5){break;}
		else if(n > maxsteps){break;}
		correction = RK5(func, y0, y0[0], h, 3);
		for(int i = 0; i < 3; i++){
			y0[i] += correction[i];
		}
		xs[n + 1] = y0[0];
		ys[n + 1] = y0[1];
		zs[n + 1] = y0[2];
		h = findH(y0, func, Xs, Ys, Zs, 1);
		phi = cartesianToSpherical(y0, 3)[1];
		if(n > 0){
			if((phi <= phiInitial && lastPhi >= phiInitial) || (phi >= phiInitial && lastPhi <= phiInitial)){break;}
		}
		lastPhi = phi;
		n += 1;
	}
}

int main(void){
    return 0;
}